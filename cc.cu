#include "hip/hip_runtime.h"
Copy/Compute Overlap Example Code

Below are two code examples for the techniques presented above, first for when the number of entries is evenly divided by the number of streams, and second, for when this is not so.
N is Evenly Divided by Number of Streams

// "Simple" version where number of entries is evenly divisible by number of streams.

// Set to a ridiculously low value to clarify mechanisms of the technique.
const uint64_t num_entries = 10;
const uint64_t num_iters = 1UL << 10;

// Allocate memory for all data entries. Make sure to pin host memory.
hipHostMalloc(&data_cpu, sizeof(uint64_t)*num_entries);
hipMalloc    (&data_gpu, sizeof(uint64_t)*num_entries);

// Set the number of streams.
const uint64_t num_streams = 2;

// Create an array of streams containing number of streams
hipStream_t streams[num_streams];
for (uint64_t stream = 0; stream < num_streams; stream++)
    hipStreamCreate(&streams[stream]);

// Set number of entries for each "chunk". Assumes `num_entries % num_streams == 0`.
const uint64_t chunk_size = num_entries / num_streams;

// For each stream, calculate indices for its chunk of full dataset and then, HtoD copy, compute, DtoH copy.
for (uint64_t stream = 0; stream < num_streams; stream++) {

    // Get start index in full dataset for this stream's work.
    const uint64_t lower = chunk_size*stream;
    
    // Stream-indexed (`data+lower`) and chunk-sized HtoD copy in the non-default stream
    // `streams[stream]`.
    hipMemcpyAsync(data_gpu+lower, data_cpu+lower, 
           sizeof(uint64_t)*chunk_size, hipMemcpyHostToDevice, 
           streams[stream]);
    
    // Stream-indexed (`data_gpu+lower`) and chunk-sized compute in the non-default stream
    // `streams[stream]`.
    decrypt_gpu<<<80*32, 64, 0, streams[stream]>>>
        (data_gpu+lower, chunk_size, num_iters);
    
    // Stream-indexed (`data+lower`) and chunk-sized DtoH copy in the non-default stream
    // `streams[stream]`.
    hipMemcpyAsync(data_cpu+lower, data_gpu+lower, 
           sizeof(uint64_t)*chunk_size, hipMemcpyDeviceToHost, 
           streams[stream]);
}

// Destroy streams.
for (uint64_t stream = 0; stream < num_streams; stream++)
    hipStreamDestroy(streams[stream]);

N is Not Evenly Divided by Number of Streams

// Able to handle when `num_entries % num_streams != 0`.

const uint64_t num_entries = 10;
const uint64_t num_iters = 1UL << 10;

hipHostMalloc(&data_cpu, sizeof(uint64_t)*num_entries);
hipMalloc    (&data_gpu, sizeof(uint64_t)*num_entries);

// Set the number of streams to not evenly divide num_entries.
const uint64_t num_streams = 3;

hipStream_t streams[num_streams];
for (uint64_t stream = 0; stream < num_streams; stream++)
    hipStreamCreate(&streams[stream]);

// Use round-up division (`sdiv`, defined in helper.cu) so `num_streams*chunk_size`
// is never less than `num_entries`.
// This can result in `num_streams*chunk_size` being greater than `num_entries`, meaning
// we will need to guard against out-of-range errors in the final "tail" stream (see below).
const uint64_t chunk_size = sdiv(num_entries, num_streams);

for (uint64_t stream = 0; stream < num_streams; stream++) {

    const uint64_t lower = chunk_size*stream;
    // For tail stream `lower+chunk_size` could be out of range, so here we guard against that.
    const uint64_t upper = min(lower+chunk_size, num_entries);
    // Since the tail stream width may not be `chunk_size`,
    // we need to calculate a separate `width` value.
    const uint64_t width = upper-lower;

    // Use `width` instead of `chunk_size`.
    hipMemcpyAsync(data_gpu+lower, data_cpu+lower, 
           sizeof(uint64_t)*width, hipMemcpyHostToDevice, 
           streams[stream]);

    // Use `width` instead of `chunk_size`.
    decrypt_gpu<<<80*32, 64, 0, streams[stream]>>>
        (data_gpu+lower, width, num_iters);

    // Use `width` instead of `chunk_size`.
    hipMemcpyAsync(data_cpu+lower, data_gpu+lower, 
           sizeof(uint64_t)*width, hipMemcpyDeviceToHost, 
           streams[stream]);
}

// Destroy streams.
for (uint64_t stream = 0; stream < num_streams; stream++)
    hipStreamDestroy(streams[stream]);

Check for Understanding